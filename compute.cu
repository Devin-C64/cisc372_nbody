#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "compute.h"
#include <hip/hip_runtime.h>

//__global__ void accelcreate(vector3** d_accels, vector3* d_values);
__global__ void pairwise( vector3* d_accels, vector3* d_hPos, double* d_mass);
__global__ void sumrows(vector3* d_accels, vector3* d_hVel, vector3* d_hPos);



//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){
	//make an acceleration matrix which is NUMENTITIES squared in size;
	/*
	vector3* values=(vector3*)malloc(sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	vector3* d_values;
	hipMalloc(&d_values, sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	hipMemcpy(d_values, values, sizeof(vector3)*NUMENTITIES*NUMENTITIES, hipMemcpyHostToDevice);
	*/
/*
	vector3** accels=(vector3**)malloc(sizeof(vector3*)*NUMENTITIES);
	for (int i=0;i<NUMENTITIES;i++)
		accels[i]=&values[i*NUMENTITIES];
	
	*/
	vector3* d_accels;
	hipMalloc(&d_accels, sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	//hipMemcpy(d_accels, accels, sizeof(vector3)*NUMENTITIES*NUMENTITIES, hipMemcpyHostToDevice);

	/*
	int accelgriddimension = (NUMENTITIES / 256) + 1;
	dim3 dimAccelGrid(accelgriddimension, 1);
	dim3 dimAccelBlock(256, 1);
	*/
	
	// accelcreate<<<dimAccelGrid,dimAccelBlock>>>(d_accels, d_values);

	double *d_mass;
	vector3 *d_hPos, *d_hVel;
	hipMalloc(&d_mass, sizeof(double) * NUMENTITIES);
	hipMemcpy(d_mass, mass, sizeof(double) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMalloc(&d_hPos, sizeof(vector3) * NUMENTITIES);
	hipMalloc(&d_hVel, sizeof(vector3) * NUMENTITIES);
	hipMemcpy(d_hPos, hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);
	hipMemcpy(d_hVel, hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyHostToDevice);

	int griddimension = (NUMENTITIES / 16) + 1;
	dim3 dimGrid(griddimension, griddimension);
	dim3 dimBlock(16, 16);

	// i = threadindex.x + blockindex.x * blockdim.x
	// j = threadindex.y + blockindex.y * blockdim.y
	// k = threadindex.z

	pairwise<<<dimGrid,dimBlock>>>(d_accels, d_hPos, d_mass);

	sumrows<<<dimGrid,dimBlock>>>(d_accels, d_hVel, d_hPos);

	hipMemcpy(hVel, d_hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hPos, d_hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);

	hipFree(d_accels);
	hipFree(d_mass);
	hipFree(d_hPos);
	hipFree(d_hVel);
}


/*
__global__ void accelcreate(vector3** d_accels, vector3* d_values){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < NUMENTITIES){
		d_accels[i]=&d_values[i*NUMENTITIES];
	}
}
*/



__global__ void pairwise( vector3* d_accels, vector3* d_hPos, double* d_mass){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int index = i * NUMENTITIES + j;
	if (i < NUMENTITIES && j < NUMENTITIES){
		if (i==j) {
			FILL_VECTOR(d_accels[index],0,0,0);
		}
			else{
				vector3 distance;
				for (int k=0;k<3;k++) distance[k]=d_hPos[i][k]-d_hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*d_mass[j]/magnitude_sq;
				FILL_VECTOR(d_accels[index],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
	}
}

__global__ void sumrows(vector3* d_accels, vector3* d_hVel, vector3* d_hPos){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < NUMENTITIES){
		vector3 accel_sum={0,0,0};
		for (int j=0;j<NUMENTITIES;j++){
			for (int k=0;k<3;k++)
				accel_sum[k]+=d_accels[i * NUMENTITIES + j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (int k=0;k<3;k++){
			d_hVel[i][k]+=accel_sum[k]*INTERVAL;
			d_hPos[i][k]+=d_hVel[i][k]*INTERVAL;
		}
	}
}
